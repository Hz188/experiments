#include "hip/hip_runtime.h"
#include <cstdio>
#include "matrix.h"
//tutorial: https://blog.csdn.net/qianqing13579/article/details/127359866
//code: https://github.com/qianqing13579/GEMM/blob/master/GEMM.cu
__global__ void NaiveGEMM(Matrix<float> A,Matrix<float> B,Matrix<float> C) {

    // 获取线程在网格内的索引
    int row = blockIdx.y * blockDim.y + threadIdx.y;// 行
    int col = blockIdx.x * blockDim.x + threadIdx.x;// 列

    // 每个线程计算矩阵C的一个元素
    if(row < C.rows && col < C.cols) {
        float c = 0;
        for (int i = 0; i < A.cols; ++i) {  //相当于一个线程，处理C中的一个元素，也就是A和B的一行/一列
            c += A(row,i) * B(i,col);// 使用A的第row行乘以B的第col列
        }
        C(row,col) = c;
    }
}



int main(int argc,char **argv)
{

    printf("GEMM_v1: NaiveGEMM\n");
    // 创建GPU A矩阵
    float *dataOfA_Device=nullptr;
    hipMalloc((void **)&dataOfA_Device, A_Host.rows*A_Host.cols*sizeof(float));
    hipMemcpy(dataOfA_Device, A_Host.data, A_Host.rows*A_Host.cols*sizeof(float), hipMemcpyHostToDevice);
    Matrix<float> A_Device(dataOfA_Device,A_Host.rows,A_Host.cols,A_Host.cols,1);

    // 创建GPU B矩阵
    float *dataOfB_Device=nullptr;
    hipMalloc((void **)&dataOfB_Device, B_Host.rows*B_Host.cols*sizeof(float));
    hipMemcpy(dataOfB_Device, B_Host.data, B_Host.rows*B_Host.cols*sizeof(float), hipMemcpyHostToDevice);
    Matrix<float> B_Device(dataOfB_Device,B_Host.rows,B_Host.cols,B_Host.cols,1);

    // 创建GPU C矩阵
    float *dataOfC_Device=nullptr;
    hipMalloc((void **)&dataOfC_Device, A_Host.rows*B_Host.cols*sizeof(float));
    Matrix<float> C_Device(dataOfC_Device,A_Host.rows,B_Host.cols,B_Host.cols,1);
    return 0;
}
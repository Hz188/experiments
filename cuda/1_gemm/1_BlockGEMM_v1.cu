#include "hip/hip_runtime.h"
#include <cstdio>
#include "matrix.h"

__global__ void BlockGEMM_V1(Matrix<float> A, Matrix<float> B, Matrix<float> C) {
    // 注意命名不要与前面的宏定义重名
    const int BLOCK_M = 16;// block的行数
    const int BLOCK_N = 16;// block的列数
    const int BLOCK_K = 16;

    // 沿着K维度循环加载一个block中对应的A和B的数据到共享内存
    float c = 0.0;
    for (int i = 0; i < A.cols / BLOCK_K; ++i) {
        // 每个block对应的全局内存中的A,B子块，即创建全局内存中A,B的view
        Matrix<float> ASub(A.data + blockIdx.y * BLOCK_M * A.strideOfRow + i * BLOCK_K, BLOCK_M, BLOCK_K, A.strideOfRow,
                           A.strideOfCol);
        Matrix<float> BSub(B.data + i * BLOCK_K * B.strideOfRow + blockIdx.x * BLOCK_N, BLOCK_K, BLOCK_N, B.strideOfRow,
                           B.strideOfCol);

        // 将Asub,BSub加载到共享内存
        // 注意：这里需要将一维线性逻辑索引转换为多维逻辑索引：startIndex->(startIndex/cols, startIndex%cols)
        __shared__ float A_Shared[BLOCK_M][BLOCK_K];
        __shared__ float B_Shared[BLOCK_K][BLOCK_N];

        int numberOfElementsPerThread = (BLOCK_K * BLOCK_M) / (blockDim.x * blockDim.y);// 每个线程需要读取多少数据
        int startIndex = numberOfElementsPerThread * (threadIdx.y * blockDim.x + threadIdx.x);// startIndex为每个线程读取的起始索引

        //搬运到shared memory
        for (int threadIndex = 0; threadIndex < numberOfElementsPerThread; ++threadIndex) {
            int logicalIndex = startIndex + threadIndex;
            A_Shared[logicalIndex / BLOCK_K][logicalIndex % BLOCK_K] = ASub(logicalIndex / BLOCK_K,
                                                                            logicalIndex % BLOCK_K);
            B_Shared[logicalIndex / BLOCK_N][logicalIndex % BLOCK_N] = BSub(logicalIndex / BLOCK_N,
                                                                            logicalIndex % BLOCK_N);
        }
        __syncthreads();

        // 每个thread计算A的一行和B的一列
        for (int k = 0; k < BLOCK_K; ++k) {
            c += A_Shared[threadIdx.y][k] * B_Shared[k][threadIdx.x];
        }
        __syncthreads();

    }

    // 将每个线程计算好的结果写回到C矩阵
    // CSub为每个线程对应的全局内存的C矩阵子块，创建C矩阵的view
    Matrix<float> CSub(C.data + (blockIdx.y * BLOCK_M * C.strideOfRow + blockIdx.x * BLOCK_N), BLOCK_M, BLOCK_N,
                       C.strideOfRow, C.strideOfCol);
    CSub(threadIdx.y, threadIdx.x) = c;

}

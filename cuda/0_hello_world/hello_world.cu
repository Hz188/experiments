
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
using namespace std;
__global__ void hello_world()
{
    printf("GPU: Hello world!\n");
}

int main(int argc,char **argv)
{
    printf("CPU: Hello world!\n");
    hello_world<<<1,10>>>();

    hipDeviceReset();//if no this line ,it can not output hello world from gpu
    return 0;
}
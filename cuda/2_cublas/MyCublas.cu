/// MyCublas.cu

#include "MyCublas.cuh"
#include "CublasUtility.h"
#include <vector>

void MatrixMulVectorCublas(
        const double* matrix, const int row, const int col,
        const double* vector, double* result
)
{
    /* step 1: create cublas handle, bind a stream */
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    double* dev_matrix = nullptr;
    double* dev_vector = nullptr;
    double* dev_result = nullptr;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&dev_matrix), sizeof(double) * row * col));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&dev_vector), sizeof(double) * col));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&dev_result), sizeof(double) * row));

    CUDA_CHECK(hipMemcpyAsync(dev_matrix, matrix, sizeof(double) * row * col, hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(dev_vector, vector, sizeof(double) * col, hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    hipblasOperation_t transa = HIPBLAS_OP_N;
    const int lda = row;
    const double alpha = 1.0;
    const double beta = 0.0;
    const int incx = 1;
    const int incy = 1;

    CUBLAS_CHECK(
            hipblasDgemv(cublasH, transa, row, col, &alpha, dev_matrix, lda, dev_vector, incx, &beta, dev_result, incy));

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(result, dev_result, sizeof(double) * row, hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /* step 5: free resources */
    CUDA_CHECK(hipFree(dev_matrix));
    CUDA_CHECK(hipFree(dev_vector));
    CUDA_CHECK(hipFree(dev_result));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());
}

